#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h> // Header for the cuFFT library

// Macro for robust CUDA error checking
#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

// Macro for robust cuFFT error checking
#define CHECK_CUFFT_ERROR(status) \
    if (status != HIPFFT_SUCCESS) { \
        std::cerr << "cuFFT Error in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

/**
 * @brief Host function to execute the FFT on the GPU using the cuFFT library.
 * This is the most optimized approach, leveraging NVIDIA's dedicated FFT algorithms.
 * @param input The input signal vector. Its size MUST be a power of 2 for best performance.
 * @return A vector containing the FFT result.
 */
std::vector<std::complex<double>> fft_gpu_cufft(const std::vector<std::complex<double>>& input) {
    hipEvent_t start_total, stop_total, start_compute, stop_compute;
    CHECK_CUDA_ERROR(hipEventCreate(&start_total));
    CHECK_CUDA_ERROR(hipEventCreate(&stop_total));
    CHECK_CUDA_ERROR(hipEventCreate(&start_compute));
    CHECK_CUDA_ERROR(hipEventCreate(&stop_compute));

    CHECK_CUDA_ERROR(hipEventRecord(start_total));

    int n = input.size();
    size_t vector_size = n * sizeof(hipDoubleComplex);

    // 1. Allocate GPU memory
    hipDoubleComplex *d_data;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_data, vector_size));

    // Convert input to cuDoubleComplex format
    std::vector<hipDoubleComplex> h_input(n);
    for (int i = 0; i < n; i++) {
        h_input[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());
    }

    // 2. Copy input vector from Host to Device
    CHECK_CUDA_ERROR(hipMemcpy(d_data, h_input.data(), vector_size, hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipEventRecord(start_compute));

    // 3. Create a cuFFT plan
    hipfftHandle plan;
    CHECK_CUFFT_ERROR(hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1)); // Z2Z: Double precision, Complex-to-Complex

    // 4. Execute the FFT
    // The transformation is done in-place, so d_data will be overwritten with the result.
    CHECK_CUFFT_ERROR(hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD));

    CHECK_CUDA_ERROR(hipEventRecord(stop_compute));

    // 5. Copy result from Device to Host
    std::vector<hipDoubleComplex> h_output(n);
    CHECK_CUDA_ERROR(hipMemcpy(h_output.data(), d_data, vector_size, hipMemcpyDeviceToHost));

    // Convert result back to std::complex
    std::vector<std::complex<double>> output(n);
    for (int i = 0; i < n; i++) {
        output[i] = { hipCreal(h_output[i]), hipCimag(h_output[i]) };
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop_total));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop_total));

    float ms_compute = 0, ms_total = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms_compute, start_compute, stop_compute));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms_total, start_total, stop_total));

    std::cout << "--- cuFFT Version ---" << std::endl;
    std::cout << "Input size N: " << n << std::endl;
    std::cout << "GPU computation time (FFT execution) (ms): " << ms_compute << std::endl;
    std::cout << "Total time (Transfers + Computation) (ms): " << ms_total << std::endl;

    // 6. Cleanup
    CHECK_CUFFT_ERROR(hipfftDestroy(plan));
    CHECK_CUDA_ERROR(hipFree(d_data));
    CHECK_CUDA_ERROR(hipEventDestroy(start_total));
    CHECK_CUDA_ERROR(hipEventDestroy(stop_total));
    CHECK_CUDA_ERROR(hipEventDestroy(start_compute));
    CHECK_CUDA_ERROR(hipEventDestroy(stop_compute));

    return output;
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>\n";
        return 1;
    }

    const char* filename = argv[1];
    std::ifstream ifs(filename);
    if (!ifs) {
        std::cerr << "Error: could not open file " << filename << "\n";
        return 1;
    }

    std::vector<std::complex<double>> data;
    double real, imag;
    while (ifs >> real >> imag) {
        data.emplace_back(real, imag);
    }
    ifs.close();

    if (data.empty()) {
        std::cerr << "Error: no data read from file " << filename << "\n";
        return 1;
    }
    
    // For best performance, FFT input size should be a power of 2.
    size_t n = data.size();
    if ((n > 0) && ((n & (n - 1)) != 0)) {
        std::cout << "Warning: Input size " << n << " is not a power of 2. cuFFT performance may be suboptimal." << std::endl;
    }


    // Execute the FFT on the GPU using cuFFT
    auto result = fft_gpu_cufft(data);

    return 0;
}
