#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <chrono>

#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " in file " << __FILE__ << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CHECK_CUFFT_ERROR(status) \
    if (status != HIPFFT_SUCCESS) { \
        std::cerr << "cuFFT Error in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

// --- Host function: FFT with cuFFT and detailed timing ---
std::vector<std::complex<double>> fft_gpu_cufft(const std::vector<std::complex<double>>& input,
                                               float& totalExecTime, float& kernelTime,
                                               float& h2dTime, float& d2hTime) {

    int n = input.size();
    size_t vector_size = n * sizeof(hipDoubleComplex);

    // Allocate device memory
    hipDoubleComplex* d_data;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, vector_size));

    // Convert input to cuDoubleComplex
    std::vector<hipDoubleComplex> h_input(n);
    for (int i = 0; i < n; i++)
        h_input[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());

    // Host->Device
    hipEvent_t startH2D, stopH2D;
    CHECK_CUDA_ERROR(hipEventCreate(&startH2D));
    CHECK_CUDA_ERROR(hipEventCreate(&stopH2D));
    CHECK_CUDA_ERROR(hipEventRecord(startH2D));
    CHECK_CUDA_ERROR(hipMemcpy(d_data, h_input.data(), vector_size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(stopH2D));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopH2D));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&h2dTime, startH2D, stopH2D));

    // Kernel (FFT) execution
    hipEvent_t startKernel, stopKernel;
    CHECK_CUDA_ERROR(hipEventCreate(&startKernel));
    CHECK_CUDA_ERROR(hipEventCreate(&stopKernel));
    CHECK_CUDA_ERROR(hipEventRecord(startKernel));

    hipfftHandle plan;
    CHECK_CUFFT_ERROR(hipfftPlan1d(&plan, n, HIPFFT_Z2Z, 1));
    CHECK_CUFFT_ERROR(hipfftExecZ2Z(plan, d_data, d_data, HIPFFT_FORWARD));

    CHECK_CUDA_ERROR(hipEventRecord(stopKernel));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopKernel));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&kernelTime, startKernel, stopKernel));

    // Device->Host
    std::vector<hipDoubleComplex> h_output(n);
    hipEvent_t startD2H, stopD2H;
    CHECK_CUDA_ERROR(hipEventCreate(&startD2H));
    CHECK_CUDA_ERROR(hipEventCreate(&stopD2H));
    CHECK_CUDA_ERROR(hipEventRecord(startD2H));
    CHECK_CUDA_ERROR(hipMemcpy(h_output.data(), d_data, vector_size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(stopD2H));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopD2H));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&d2hTime, startD2H, stopD2H));

    // Convert to std::complex
    std::vector<std::complex<double>> output(n);
    for (int i = 0; i < n; i++)
        output[i] = { hipCreal(h_output[i]), hipCimag(h_output[i]) };

    // Total execution time
    totalExecTime = h2dTime + kernelTime + d2hTime;

    // Cleanup
    CHECK_CUFFT_ERROR(hipfftDestroy(plan));
    CHECK_CUDA_ERROR(hipFree(d_data));
    hipEventDestroy(startH2D); hipEventDestroy(stopH2D);
    hipEventDestroy(startKernel); hipEventDestroy(stopKernel);
    hipEventDestroy(startD2H); hipEventDestroy(stopD2H);

    return output;
}

// --- MAIN ---
int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <threads_per_block> <input_file> [num_runs]\n";
        return 1;
    }

    int threadsPerBlock = std::max(1, std::stoi(argv[1])); // Not used in this algorithm but kept for compatibility with other implementations
    const char* filename = argv[2];
    int num_runs = (argc >= 4) ? std::max(1, std::stoi(argv[3])) : 1;

    // File reading
    auto t_start = std::chrono::high_resolution_clock::now();
    std::ifstream ifs(filename);
    if (!ifs) { std::cerr << "Error opening file.\n"; return 1; }

    std::vector<std::complex<double>> data;
    double re, im;
    while (ifs >> re >> im)
        data.emplace_back(re, im);
    ifs.close();
    auto t_end = std::chrono::high_resolution_clock::now();
    float readTime = std::chrono::duration<float, std::milli>(t_end - t_start).count();
    std::cout << "[RESULTS] ReadingTime: " << readTime << "ms\n";

    // Run multiple times
    float totalTimeSum = 0;
    for (int run = 1; run <= num_runs; ++run) {
        float totalExec = 0, kernelTime = 0, h2dTime = 0, d2hTime = 0;
        auto result = fft_gpu_cufft(data, totalExec, kernelTime, h2dTime, d2hTime);

        std::cout << "[RESULTS] ExecutionTime(run=" << run << "): " << totalExec << "ms\n";
        std::cout << "  (Details) Host->Device: " << h2dTime << "ms\n";
        std::cout << "  (Details) Kernel: " << kernelTime << "ms\n";
        std::cout << "  (Details) Device->Host: " << d2hTime << "ms\n";

        totalTimeSum += totalExec;
    }

    std::cout << "[RESULTS] TotalTime: " << (readTime + totalTimeSum) << "ms\n";

    return 0;
}