#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <chrono>
#include <iomanip>

using namespace std;

// --- CUDA error checking macro ---
#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        cerr << "CUDA Error: " << hipGetErrorString(err) \
             << " in file " << __FILE__ << " at line " << __LINE__ << endl; \
        exit(EXIT_FAILURE); \
    }

// --- Constant PI in GPU memory ---
__constant__ double d_PI;

// --- Kernel: performs one FFT stage ---
__global__ void fft_stage(hipFloatComplex* data, int n, int step) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int halfStep = step / 2;

    if (tid < n / 2) {
        int i = (tid / halfStep) * step + (tid % halfStep);

        hipFloatComplex u = data[i];
        hipFloatComplex v = data[i + halfStep];

        float angle = -2.0f * d_PI * (tid % halfStep) / step;
        hipFloatComplex w = make_hipFloatComplex(cosf(angle), sinf(angle));

        hipFloatComplex t = make_hipFloatComplex(
            v.x * w.x - v.y * w.y,
            v.x * w.y + v.y * w.x
        );

        data[i]            = make_hipFloatComplex(u.x + t.x, u.y + t.y);
        data[i + halfStep] = make_hipFloatComplex(u.x - t.x, u.y - t.y);
    }
}

// --- Host function: FFT GPU with detailed timing ---
void fft_gpu(vector<complex<float>>& input, int threadsPerBlock,
             float& totalExecTime, float& kernelTime,
             float& h2dTime, float& d2hTime) {

    const double h_PI = acos(-1.0);
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_PI), &h_PI, sizeof(double)));

    int N = input.size();
    size_t size = N * sizeof(hipFloatComplex);

    // Allocate device memory
    hipFloatComplex* d_data;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, size));

    // Host->Device copy
    hipEvent_t startH2D, stopH2D;
    CHECK_CUDA_ERROR(hipEventCreate(&startH2D));
    CHECK_CUDA_ERROR(hipEventCreate(&stopH2D));
    CHECK_CUDA_ERROR(hipEventRecord(startH2D));
    CHECK_CUDA_ERROR(hipMemcpy(d_data, input.data(), size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(stopH2D));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopH2D));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&h2dTime, startH2D, stopH2D));

    // Kernel execution
    hipEvent_t startKernel, stopKernel;
    CHECK_CUDA_ERROR(hipEventCreate(&startKernel));
    CHECK_CUDA_ERROR(hipEventCreate(&stopKernel));
    CHECK_CUDA_ERROR(hipEventRecord(startKernel));

    for (int step = 2; step <= N; step <<= 1) {
        int butterflies = N / 2;
        int blocks = (butterflies + threadsPerBlock - 1) / threadsPerBlock;
        fft_stage<<<blocks, threadsPerBlock>>>(d_data, N, step);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    }

    CHECK_CUDA_ERROR(hipEventRecord(stopKernel));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopKernel));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&kernelTime, startKernel, stopKernel));

    // Device->Host copy
    vector<hipFloatComplex> h_output(N);
    hipEvent_t startD2H, stopD2H;
    CHECK_CUDA_ERROR(hipEventCreate(&startD2H));
    CHECK_CUDA_ERROR(hipEventCreate(&stopD2H));
    CHECK_CUDA_ERROR(hipEventRecord(startD2H));
    CHECK_CUDA_ERROR(hipMemcpy(h_output.data(), d_data, size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(stopD2H));
    CHECK_CUDA_ERROR(hipEventSynchronize(stopD2H));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&d2hTime, startD2H, stopD2H));

    // Convert back to std::complex
    for (int i = 0; i < N; i++) {
        input[i] = { h_output[i].x, h_output[i].y };
    }

    // Total execution
    totalExecTime = h2dTime + kernelTime + d2hTime;

    // Cleanup
    CHECK_CUDA_ERROR(hipFree(d_data));
    CHECK_CUDA_ERROR(hipEventDestroy(startH2D));
    CHECK_CUDA_ERROR(hipEventDestroy(stopH2D));
    CHECK_CUDA_ERROR(hipEventDestroy(startKernel));
    CHECK_CUDA_ERROR(hipEventDestroy(stopKernel));
    CHECK_CUDA_ERROR(hipEventDestroy(startD2H));
    CHECK_CUDA_ERROR(hipEventDestroy(stopD2H));
}

// --- MAIN ---
int main(int argc, char* argv[]) {
    if (argc < 3) {
        cerr << "Usage: " << argv[0] << " <threads_per_block> <input_file> [num_runs]\n";
        return 1;
    }

    int threadsPerBlock = stoi(argv[1]);
    const char* filename = argv[2];
    int numRuns = (argc >= 4) ? std::max(1, std::stoi(argv[3])) : 1; // default 1

    // Measure file reading time
    auto startRead = chrono::high_resolution_clock::now();
    ifstream ifs(filename);
    if (!ifs) { cerr << "Error opening file.\n"; return 1; }

    vector<complex<float>> data;
    float re, im;
    while (ifs >> re >> im) data.emplace_back(re, im);
    ifs.close();
    auto endRead = chrono::high_resolution_clock::now();
    float readTime = chrono::duration<float, milli>(endRead - startRead).count();

    cout << "[RESULTS] ReadingTime: " << readTime << "ms\n";

    if ((data.size() & (data.size() - 1)) != 0) {
        cerr << "FFT requires power-of-2 size.\n";
        return 1;
    }

    // Run FFT multiple times according to numRuns
    float totalTimeSum = 0;
    for (int run = 1; run <= numRuns; ++run) {
        float totalExec = 0, kernelTime = 0, h2dTime = 0, d2hTime = 0;
        fft_gpu(data, threadsPerBlock, totalExec, kernelTime, h2dTime, d2hTime);

        cout << "[RESULTS] ExecutionTime(run=" << run << "): " << totalExec << "ms\n";
        cout << "  (Details) Host->Device: " << h2dTime << "ms\n";
        cout << "  (Details) Kernel: " << kernelTime << "ms\n";
        cout << "  (Details) Device->Host: " << d2hTime << "ms\n";

        totalTimeSum += totalExec;
    }

    cout << "[RESULTS] TotalTime: " << (readTime + totalTimeSum) << "ms\n";

    return 0;
}