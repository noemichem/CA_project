#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <chrono>
#include <hip/hip_math_constants.h>

// --- CUDA Error Checking Macro ---
#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " in file " << __FILE__ << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

// ------------------- GPU Utility Ottimizzata -------------------
__device__ __forceinline__ hipFloatComplex complex_mul(hipFloatComplex a, hipFloatComplex b) {
    // (a.x + i*a.y) * (b.x + i*b.y) = (a.x*b.x - a.y*b.y) + i*(a.x*b.y + a.y*b.x)
    float real = __fmaf_rn(-a.y, b.y, __fmul_rn(a.x, b.x));
    float imag = __fmaf_rn(a.y, b.x, __fmul_rn(a.x, b.y));
    return make_hipFloatComplex(real, imag);
}

__device__ __forceinline__ hipFloatComplex complex_add(hipFloatComplex a, hipFloatComplex b) {
    return make_hipFloatComplex(__fadd_rn(a.x, b.x), __fadd_rn(a.y, b.y));
}

__device__ __forceinline__ hipFloatComplex complex_sub(hipFloatComplex a, hipFloatComplex b) {
    return make_hipFloatComplex(__fsub_rn(a.x, b.x), __fsub_rn(a.y, b.y));
}

// ------------------- Bit Reversal -------------------
__global__ void bit_reverse_kernel(hipFloatComplex* data, size_t n, int logn) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    unsigned long long idx64 = static_cast<unsigned long long>(i);
    unsigned long long r64   = __brevll(idx64);
    size_t rev = static_cast<size_t>(r64 >> (64 - logn));

    if (i < rev) {
        hipFloatComplex tmp = data[i];
        data[i]  = data[rev];
        data[rev] = tmp;
    }
}

// ------------------- FFT Kernel Ottimizzato con Shared Memory Sicura -------------------
__global__ void fft_kernel(hipFloatComplex* data, size_t n, int logn) {
    extern __shared__ hipFloatComplex s_data[];

    size_t tid = threadIdx.x;
    size_t gid = blockIdx.x * blockDim.x + tid;
    if (gid >= n) return;

    // Carica dati in shared memory solo se block può contenere tutta la fase
    for (int s = 1; s <= logn; ++s) {
        size_t m = 1ULL << s;
        size_t half = m >> 1;

        if (m <= blockDim.x) {
            // Shared memory FFT
            if (tid < m) s_data[tid] = data[blockIdx.x * blockDim.x + tid];
            __syncthreads();

            size_t j = tid & (half - 1);
            size_t block_idx = (tid >> (s - 1)) * m;
            size_t index1 = block_idx + j;
            size_t index2 = index1 + half;

            if (index2 < n) {
                float j_f = static_cast<float>(j);
                float m_f = static_cast<float>(m);
                float angle = __fdividef(__fmul_rn(-2.0f * HIP_PI_F, j_f), m_f);

                float s_val, c_val;
                __sincosf(angle, &s_val, &c_val);
                hipFloatComplex w = make_hipFloatComplex(c_val, s_val);

                hipFloatComplex u = s_data[index1];
                hipFloatComplex v = complex_mul(s_data[index2], w);

                s_data[index1] = complex_add(u, v);
                s_data[index2] = complex_sub(u, v);
            }
            __syncthreads();

            // Scrivi fase su memoria globale
            if (tid < m) data[blockIdx.x * blockDim.x + tid] = s_data[tid];

        } else {
            // FFT su memoria globale
            size_t j = tid & (half - 1);
            size_t block_idx = (tid >> (s - 1)) * m;
            size_t index1 = block_idx + j;
            size_t index2 = index1 + half;

            if (index2 < n) {
                float j_f = static_cast<float>(j);
                float m_f = static_cast<float>(m);
                float angle = __fdividef(__fmul_rn(-2.0f * HIP_PI_F, j_f), m_f);

                float s_val, c_val;
                __sincosf(angle, &s_val, &c_val);
                hipFloatComplex w = make_hipFloatComplex(c_val, s_val);

                hipFloatComplex u = data[index1];
                hipFloatComplex v = complex_mul(data[index2], w);

                data[index1] = complex_add(u, v);
                data[index2] = complex_sub(u, v);
            }
            __syncthreads(); // sicuro per i thread nello stesso block
        }
    }
}

// ------------------- Host Wrapper -------------------
void fft_gpu(std::vector<std::complex<float>>& input, int threadsPerBlock,
             float& totalExecTime, float& kernelTime,
             float& h2dTime, float& d2hTime) {

    size_t n = input.size();
    int logn = 0;
    while ((1ULL << logn) < n) ++logn;

    size_t sizeBytes = n * sizeof(hipFloatComplex);

    hipFloatComplex* d_data = nullptr;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, sizeBytes));

    // Convert input to cuFloatComplex
    std::vector<hipFloatComplex> h_cdata(n);
    for (size_t i = 0; i < n; ++i) {
        h_cdata[i] = make_hipFloatComplex(input[i].real(), input[i].imag());
    }

    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    // Host -> Device
    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy(d_data, h_cdata.data(), sizeBytes, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&h2dTime, start, stop));

    // Kernel execution
    dim3 block(threadsPerBlock);
    dim3 grid((n + block.x - 1) / block.x);
    size_t shared_mem_size = block.x * sizeof(hipFloatComplex);

    CHECK_CUDA_ERROR(hipEventRecord(start));
    bit_reverse_kernel<<<grid, block>>>(d_data, n, logn);
    hipDeviceSynchronize();
    fft_kernel<<<grid, block, shared_mem_size>>>(d_data, n, logn);
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&kernelTime, start, stop));

    // Device -> Host
    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy(h_cdata.data(), d_data, sizeBytes, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&d2hTime, start, stop));

    // Convert back to std::complex
    for (size_t i = 0; i < n; ++i) {
        input[i] = { h_cdata[i].x, h_cdata[i].y };
    }

    totalExecTime = h2dTime + kernelTime + d2hTime;

    CHECK_CUDA_ERROR(hipFree(d_data));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));
}

// ------------------- MAIN -------------------
int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <threads_per_block> <input_file> [num_runs]\n";
        return 1;
    }

    int threadsPerBlock = std::stoi(argv[1]);
    const char* filename = argv[2];
    int numRuns = (argc >= 4) ? std::max(1, std::stoi(argv[3])) : 1;

    auto startRead = std::chrono::high_resolution_clock::now();
    std::ifstream ifs(filename);
    if (!ifs) { std::cerr << "Error opening file.\n"; return 1; }

    std::vector<std::complex<float>> data;
    float re, im;
    while (ifs >> re >> im) data.emplace_back(re, im);
    ifs.close();
    auto endRead = std::chrono::high_resolution_clock::now();
    float readTime = std::chrono::duration<float, std::milli>(endRead - startRead).count();
    std::cout << "[RESULTS] ReadingTime: " << readTime << "ms\n";

    if (data.empty() || (data.size() & (data.size() - 1)) != 0) {
        std::cerr << "Input data size must be a power-of-2.\n";
        return 1;
    }

    float totalTimeSum = 0;
    for (int run = 1; run <= numRuns; ++run) {
        float totalExec = 0, kernelTime = 0, h2dTime = 0, d2hTime = 0;
        std::vector<std::complex<float>> inputCopy = data;
        fft_gpu(inputCopy, threadsPerBlock, totalExec, kernelTime, h2dTime, d2hTime);

        std::cout << "[RESULTS] ExecutionTime(run=" << run << "): " << totalExec << "ms\n";
        std::cout << "  (Details) Host->Device: " << h2dTime << "ms\n";
        std::cout << "  (Details) Kernel: " << kernelTime << "ms\n";
        std::cout << "  (Details) Device->Host: " << d2hTime << "ms\n";

        totalTimeSum += totalExec;
    }

    std::cout << "[RESULTS] TotalTime: " << (readTime + totalTimeSum) << "ms\n";
    if (numRuns > 1) {
        std::cout << "[RESULTS] AverageExecutionTime: " << (totalTimeSum / numRuns) << "ms\n";
    }

    return 0;
}