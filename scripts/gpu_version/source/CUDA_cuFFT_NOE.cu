// miglioramenti: float 32, pinned memory, stream, batch FFT, da testare

#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <chrono>

#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " in file " << __FILE__ << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CHECK_CUFFT_ERROR(status) \
    if (status != HIPFFT_SUCCESS) { \
        std::cerr << "cuFFT Error in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

// --- Host function: FFT with cuFFT (optimized) ---
std::vector<std::complex<float>> fft_gpu_cufft(
        const std::vector<std::complex<float>>& input,
        int batch, float& totalExecTime, float& kernelTime,
        float& h2dTime, float& d2hTime) {

    int n = input.size();  
    size_t vector_size = n * batch * sizeof(hipfftComplex);

    // ⚡ 1. Allocazione Host in **Pinned Memory** (più veloce per trasferimenti H2D/D2H)
    hipfftComplex* h_input;
    hipfftComplex* h_output;
    CHECK_CUDA_ERROR(hipHostMalloc(&h_input, vector_size, hipHostMallocDefault, hipHostMallocDefault));  // pinned
    CHECK_CUDA_ERROR(hipHostMalloc(&h_output, vector_size, hipHostMallocDefault, hipHostMallocDefault));

    // Riempimento dati batchati (replico lo stesso input per semplicità)
    for (int b = 0; b < batch; b++) {
        for (int i = 0; i < n; i++) {
            int idx = b * n + i;
            h_input[idx].x = input[i].real();
            h_input[idx].y = input[i].imag();
        }
    }

    // ⚡ 2. Allocazione memoria Device
    hipfftComplex* d_data;
    CHECK_CUDA_ERROR(hipMalloc(&d_data, vector_size));

    // ⚡ 3. Creazione Stream per overlap
    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    // ⚡ 4. Creazione Piano cuFFT con **batch**
    hipfftHandle plan;
    CHECK_CUFFT_ERROR(hipfftPlan1d(&plan, n, HIPFFT_C2C, batch));
    CHECK_CUFFT_ERROR(hipfftSetStream(plan, stream));  // lega il piano allo stream

    // Timing
    hipEvent_t startH2D, stopH2D, startKernel, stopKernel, startD2H, stopD2H;
    hipEventCreate(&startH2D); hipEventCreate(&stopH2D);
    hipEventCreate(&startKernel); hipEventCreate(&stopKernel);
    hipEventCreate(&startD2H); hipEventCreate(&stopD2H);

    // --- Host->Device (asincrono, pinned + stream)
    hipEventRecord(startH2D, stream);
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_data, h_input, vector_size,
                                     hipMemcpyHostToDevice, stream));
    hipEventRecord(stopH2D, stream);

    // --- FFT execution (in stream, su batch intero)
    hipEventRecord(startKernel, stream);
    CHECK_CUFFT_ERROR(hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD));
    hipEventRecord(stopKernel, stream);

    // --- Device->Host (asincrono, pinned + stream)
    hipEventRecord(startD2H, stream);
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_data, vector_size,
                                     hipMemcpyDeviceToHost, stream));
    hipEventRecord(stopD2H, stream);

    // Sync finale
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    // Calcolo tempi
    hipEventElapsedTime(&h2dTime, startH2D, stopH2D);
    hipEventElapsedTime(&kernelTime, startKernel, stopKernel);
    hipEventElapsedTime(&d2hTime, startD2H, stopD2H);
    totalExecTime = h2dTime + kernelTime + d2hTime;

    // Conversione output in std::complex<float>
    std::vector<std::complex<float>> output(n * batch);
    for (int i = 0; i < n * batch; i++)
        output[i] = { h_output[i].x, h_output[i].y };

    // Cleanup
    hipfftDestroy(plan);
    hipFree(d_data);
    hipHostFree(h_input);
    hipHostFree(h_output);
    hipStreamDestroy(stream);

    return output;
}

// --- MAIN ---
int main(int argc, char* argv[]) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <threads_per_block> <input_file> [num_runs]\n";
        return 1;
    }

    int batch = std::max(1, std::stoi(argv[1]));
    const char* filename = argv[2];
    int num_runs = (argc >= 4) ? std::max(1, std::stoi(argv[3])) : 1;

    // File reading
    auto t_start = std::chrono::high_resolution_clock::now();
    std::ifstream ifs(filename);
    if (!ifs) { std::cerr << "Error opening file.\n"; return 1; }

    std::vector<std::complex<float>> data;
    float re, im;
    while (ifs >> re >> im)
        data.emplace_back(re, im);
    ifs.close();
    auto t_end = std::chrono::high_resolution_clock::now();
    float readTime = std::chrono::duration<float, std::milli>(t_end - t_start).count();
    std::cout << "[RESULTS] ReadingTime: " << readTime << "ms\n";

    // Run multiple times
    float totalTimeSum = 0;
    for (int run = 1; run <= num_runs; ++run) {
        float totalExec = 0, kernelTime = 0, h2dTime = 0, d2hTime = 0;
        auto result = fft_gpu_cufft(data, batch, totalExec, kernelTime, h2dTime, d2hTime);

        std::cout << "[RESULTS] ExecutionTime(run=" << run << "): " << totalExec << "ms\n";
        std::cout << "  (Details) Host->Device: " << h2dTime << "ms\n";
        std::cout << "  (Details) Kernel: " << kernelTime << "ms\n";
        std::cout << "  (Details) Device->Host: " << d2hTime << "ms\n";

        totalTimeSum += totalExec;
    }

    std::cout << "[RESULTS] TotalTime: " << (readTime + totalTimeSum) << "ms\n";

    return 0;
}
