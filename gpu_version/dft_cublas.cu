#include <iostream>
#include <fstream>
#include <vector>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h> // Header for cuBLAS

// Macro for robust CUDA error checking
#define CHECK_CUDA_ERROR(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) \
                  << " in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

// Macro for robust cuBLAS error checking
#define CHECK_CUBLAS_ERROR(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error in file " << __FILE__ \
                  << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    }

const double PI = acos(-1.0);

/**
 * @brief Kernel to generate the N x N DFT matrix directly on the GPU.
 * Each thread computes one element of the matrix.
 * @param dft_matrix The output matrix W.
 * @param n The dimension of the matrix (N).
 */
__global__ void generate_dft_matrix_kernel(hipDoubleComplex* dft_matrix, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // k
    int col = blockIdx.x * blockDim.x + threadIdx.x; // t

    if (row < n && col < n) {
        double angle = -2.0 * PI * row * col / n;
        dft_matrix[row * n + col] = make_hipDoubleComplex(cos(angle), sin(angle));
    }
}

/**
 * @brief Host function to execute the DFT on the GPU using cuBLAS.
 * This version generates the DFT matrix on the GPU and then uses
 * cuBLAS for a highly optimized matrix-vector multiplication.
 * @param input The input signal vector.
 * @return A vector containing the DFT result.
 */
std::vector<std::complex<double>> dft_gpu_cublas(const std::vector<std::complex<double>>& input) {
    hipEvent_t start_total, stop_total, start_compute, stop_compute;
    CHECK_CUDA_ERROR(hipEventCreate(&start_total));
    CHECK_CUDA_ERROR(hipEventCreate(&stop_total));
    CHECK_CUDA_ERROR(hipEventCreate(&start_compute));
    CHECK_CUDA_ERROR(hipEventCreate(&stop_compute));

    CHECK_CUDA_ERROR(hipEventRecord(start_total));

    int n = input.size();
    size_t vector_size = n * sizeof(hipDoubleComplex);
    size_t matrix_size = n * n * sizeof(hipDoubleComplex);

    // 1. Initialize cuBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS_ERROR(hipblasCreate(&handle));

    // 2. Allocate GPU memory
    hipDoubleComplex *d_input, *d_output, *d_dft_matrix;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_input, vector_size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, vector_size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_dft_matrix, matrix_size));

    // Convert input to cuDoubleComplex format
    std::vector<hipDoubleComplex> h_input(n);
    for (int i = 0; i < n; i++) {
        h_input[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());
    }

    // 3. Copy input vector from Host to Device
    CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input.data(), vector_size, hipMemcpyHostToDevice));

    CHECK_CUDA_ERROR(hipEventRecord(start_compute));

    // 4. Generate DFT Matrix on the GPU
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (n + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (n + threadsPerBlock.y - 1) / threadsPerBlock.y
    );
    generate_dft_matrix_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_dft_matrix, n);
    CHECK_CUDA_ERROR(hipGetLastError());

    // 5. Perform Matrix-Vector Multiplication using cuBLAS: d_output = d_dft_matrix * d_input
    const hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0); // alpha = 1
    const hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);  // beta = 0
    
    // cublasZgemv performs Y = alpha*A*x + beta*Y
    CHECK_CUBLAS_ERROR(hipblasZgemv(handle, HIPBLAS_OP_N, n, n, &alpha, d_dft_matrix, n, d_input, 1, &beta, d_output, 1));

    CHECK_CUDA_ERROR(hipEventRecord(stop_compute));

    // 6. Copy result from Device to Host
    std::vector<hipDoubleComplex> h_output(n);
    CHECK_CUDA_ERROR(hipMemcpy(h_output.data(), d_output, vector_size, hipMemcpyDeviceToHost));

    // Convert result back to std::complex
    std::vector<std::complex<double>> output(n);
    for (int i = 0; i < n; i++) {
        output[i] = { hipCreal(h_output[i]), hipCimag(h_output[i]) };
    }

    CHECK_CUDA_ERROR(hipEventRecord(stop_total));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop_total));

    float ms_compute = 0, ms_total = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms_compute, start_compute, stop_compute));
    CHECK_CUDA_ERROR(hipEventElapsedTime(&ms_total, start_total, stop_total));

    std::cout << "--- cuBLAS Version ---" << std::endl;
    std::cout << "Input size N: " << n << std::endl;
    std::cout << "GPU computation time (Matrix Gen + GEMV) (ms): " << ms_compute << std::endl;
    std::cout << "Total time (Transfers + Computation) (ms): " << ms_total << std::endl;

    // 7. Cleanup
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipFree(d_dft_matrix));
    CHECK_CUBLAS_ERROR(hipblasDestroy(handle));
    CHECK_CUDA_ERROR(hipEventDestroy(start_total));
    CHECK_CUDA_ERROR(hipEventDestroy(stop_total));
    CHECK_CUDA_ERROR(hipEventDestroy(start_compute));
    CHECK_CUDA_ERROR(hipEventDestroy(stop_compute));

    return output;
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <input_file>\n";
        return 1;
    }

    const char* filename = argv[1];
    std::ifstream ifs(filename);
    if (!ifs) {
        std::cerr << "Error: could not open file " << filename << "\n";
        return 1;
    }

    std::vector<std::complex<double>> data;
    double real, imag;
    while (ifs >> real >> imag) {
        data.emplace_back(real, imag);
    }
    ifs.close();

    if (data.empty()) {
        std::cerr << "Error: no data read from file " << filename << "\n";
        return 1;
    }

    // Execute the DFT on the GPU using cuBLAS
    auto result = dft_gpu_cublas(data);

    return 0;
}
